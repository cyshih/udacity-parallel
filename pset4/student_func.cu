#include "hip/hip_runtime.h"
//Udacity HW 4
//Radix Sorting

#include "utils.h"
#include <thrust/host_vector.h>

/* Red Eye Removal
   ===============
   
   For this assignment we are implementing red eye removal.  This is
   accomplished by first creating a score for every pixel that tells us how
   likely it is to be a red eye pixel.  We have already done this for you - you
   are receiving the scores and need to sort them in ascending order so that we
   know which pixels to alter to remove the red eye.

   Note: ascending order == smallest to largest

   Each score is associated with a position, when you sort the scores, you must
   also move the positions accordingly.

   Implementing Parallel Radix Sort with CUDA
   ==========================================

   The basic idea is to construct a histogram on each pass of how many of each
   "digit" there are.   Then we scan this histogram so that we know where to put
   the output of each digit.  For example, the first 1 must come after all the
   0s so we have to know how many 0s there are to be able to start moving 1s
   into the correct position.

   1) Histogram of the number of occurrences of each digit
   2) Exclusive Prefix Sum of Histogram
   3) Determine relative offset of each digit
        For example [0 0 1 1 0 0 1]
                ->  [0 1 0 1 2 3 2]
   4) Combine the results of steps 2 & 3 to determine the final
      output location for each element and move it there

   LSB Radix sort is an out-of-place sort and you will need to ping-pong values
   between the input and output buffers we have provided.  Make sure the final
   sorted results end up in the output buffer!  Hint: You may need to do a copy
   at the end.

 */

__global__ void compute_histogram(unsigned int* const input,
                             unsigned int* output,
                             unsigned int mask,
                             unsigned int iteration,
                             const size_t numElems) {

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= numElems) {
        return;
    }
    unsigned int bin = (input[idx] & mask) >> iteration;
    atomicAdd(&(output[1-bin]), 1);
}

__global__ void compute_cumulative_hist_naive(unsigned int* d_in,
                                        unsigned int* d_out,
                                        const size_t numBins) {
    const int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= numBins) {
        return;
    }

    extern __shared__ unsigned int tmp[];
    int pout = 0, pin = 1;

    tmp[pout * numBins + idx] = (idx > 0)? d_in[idx - 1] : 0;
    __syncthreads();

    for (int offset = 1; offset < numBins; offset *= 2) {
        pout = 1 - pout;
        pin = 1 - pout;
        if (idx >= offset) {
            tmp[pout * numBins + idx] += tmp[pin * numBins + idx - offset];
        } else {
            tmp[pout * numBins + idx] += tmp[pin * numBins + idx];
        }
        __syncthreads();
    }

    d_out[idx] = tmp[pout * numBins + idx];        
}

__global__ void scatter(unsigned int* const d_inputVals,
                        unsigned int* const d_inputPos,
                        unsigned int* const d_outputVals,
                        unsigned int* const d_outputPos,
                        unsigned int* cum_hist,
                        const size_t numElems,
                        const size_t numBins,
                        unsigned int mask,
                        unsigned int iteration) {

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= numElems) {
        return;
    }
    
    unsigned int bin = (d_inputVals[idx] & mask) >> iteration;
    unsigned int last_elem_bin = (d_inputVals[numElems - 1] & mask) >> iteration;
    unsigned int total_zeros = cum_hist[numBins - 1] + 1 - last_elem_bin;
    unsigned int t = idx - cum_hist[idx] + total_zeros;

    unsigned int dst_pos = bin? t : cum_hist[idx];

    d_outputVals[dst_pos] = d_inputVals[idx];
    d_outputPos[dst_pos] = d_inputPos[idx];
}

__global__ void complete_one_sort(unsigned int* const d_inputVals,
                                  unsigned int* const d_inputPos,
                                  unsigned int* const d_outputVals,
                                  unsigned int* const d_outputPos,
                                  const size_t numElems,
                                  unsigned int* offset,
                                  unsigned int* cum_hist,
                                  unsigned int mask,
                                  unsigned int iteration) {

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= numElems) {
        return;
    }

    unsigned int bin = (d_inputVals[idx] & mask) >> iteration;

    unsigned int dst_pos = offset[idx]; 
    d_outputVals[dst_pos] = d_inputVals[idx];
    d_outputPos[dst_pos] = d_inputPos[idx];
}

void your_sort(unsigned int* const d_inputVals,
               unsigned int* const d_inputPos,
               unsigned int* const d_outputVals,
               unsigned int* const d_outputPos,
               const size_t numElems)
{ 
    const int numBits = 1;
    const int numBins = 1 << numBits;

    // Initialize histogram and cumulative histogram memory
    unsigned int* hist;
    unsigned int* cum_hist;
    /*unsigned int* offset;*/
    checkCudaErrors(hipMalloc(&hist, sizeof(unsigned int) * numBins));
    checkCudaErrors(hipMalloc(&cum_hist, sizeof(unsigned int) * numBins));
    /*checkCudaErrors(hipMalloc(offset, sizeof(unsigned int) * numElems));*/

    // Compute block and grid size
    const dim3 hist_blockSize(256, 1, 1);
    const dim3 hist_gridSize((numElems + 256 - 1)/256, 1, 1);
    const dim3 scan_blockSize(1, 1, 1);
    const dim3 scan_gridSize(numBins, 1, 1);
    const dim3 offset_blockSize(256, 1, 1);
    const dim3 offset_gridSize((numElems + 256 - 1)/256, 1, 1);
    const dim3 sort_blockSize(256, 1, 1);
    const dim3 sort_gridSize((numElems + 256 - 1)/256, 1, 1);

    unsigned int *tmp;

    unsigned int *vals_src;
    unsigned int *pos_src;
    unsigned int *vals_dst;
    unsigned int *pos_dst;

    checkCudaErrors(hipMalloc(&vals_src, sizeof(unsigned int) * numElems));
    checkCudaErrors(hipMalloc(&pos_src, sizeof(unsigned int) * numElems));
    checkCudaErrors(hipMalloc(&vals_dst, sizeof(unsigned int) * numElems));
    checkCudaErrors(hipMalloc(&pos_dst, sizeof(unsigned int) * numElems));
    checkCudaErrors(hipMalloc(&tmp, sizeof(unsigned int) * numElems));

    checkCudaErrors(hipMemcpy(vals_src, d_inputVals, sizeof(unsigned int) * numElems, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(pos_src, d_inputPos, sizeof(unsigned int) * numElems, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(vals_dst, d_outputVals, sizeof(unsigned int) * numElems, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(pos_dst, d_outputPos, sizeof(unsigned int) * numElems, hipMemcpyDeviceToDevice));

    for (unsigned int i = 0; i < 8 * sizeof(unsigned int); i+= numBits) {
        unsigned int mask = (numBins - 1) << i;
        checkCudaErrors(hipMemset(hist, 0, sizeof(unsigned int) * numBins));
        checkCudaErrors(hipMemset(cum_hist, 0, sizeof(unsigned int) * numBins));

        // create histogram of number of occurrences of each digit
        compute_histogram<<<hist_gridSize, hist_blockSize>>>(vals_src, hist, mask, i, numElems);
        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

        // exclusive prefix sum of histogram
        compute_cumulative_hist_naive<<<scan_gridSize, scan_blockSize, sizeof(unsigned int) * numBins>>>(hist, cum_hist, numBins);
        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

        // determine relative offset of each digit
        /*checkCudaErrors(hipMemset(offset, 0, sizeof(unsigned int) * numElems));*/
        /*compute_relative_offset<<<offset_gridSize, offset_blockSize>>>(d_inputVals,*/
        /*                                                               offset,*/
        /*                                                               cum_hist,*/
        /*                                                               numElems,*/
        /*                                                               mask,*/
        /*                                                               iteration);*/
        /*hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());*/

        scatter<<<sort_gridSize, sort_blockSize>>>(vals_src,
                                                   pos_src,
                                                   vals_dst,
                                                   pos_dst,
                                                   cum_hist,
                                                   numElems,
                                                   numBins,
                                                   mask,
                                                   i);
        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
        
        // Swap input and output
        tmp = vals_dst;
        vals_dst = vals_src;
        vals_src = tmp;

        tmp = pos_dst;
        pos_dst = pos_src;
        pos_src = tmp;
    }

    // Swap input and output
    checkCudaErrors(hipMemcpy(d_outputVals, d_inputVals, sizeof(unsigned int) * numElems, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(d_outputPos, d_inputPos, sizeof(unsigned int) * numElems, hipMemcpyDeviceToDevice));

    // Free memory
    checkCudaErrors(hipFree(hist));
    checkCudaErrors(hipFree(cum_hist));
}
